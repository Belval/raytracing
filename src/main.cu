#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hiprand/hiprand_kernel.h>

#include "bvh_node.cuh"
#include "camera.cuh"
#include "entitylist.cuh"
#include "float.h"
#include "sphere.cuh"
#include "rect.cuh"
#include "diffuse_light.cuh"
#include "moving_sphere.cuh"
#include "lambertian.cuh"
#include "metal.cuh"
#include "transparent.cuh"
#include "texture.cuh"
#include "box.cuh"
#include "transform.cuh"
#include "isotropic.cuh"
#include "constant_medium.cuh"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ Vec3 color(const Ray& r, const Vec3& background, Entity **world, hiprandState *local_rand_state) {
    Ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1.0, 1.0, 1.0);
    Vec3 cur_emitted = Vec3(0.0, 0.0, 0.0);
    for(int i = 0; i < 100; i++) {
        HitRecord rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            Ray scattered;
            Vec3 attenuation;
            Vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_emitted += emitted * cur_attenuation;
                cur_ray = scattered;
            }
            else {
                return cur_emitted + emitted * cur_attenuation;
            }
        }
        else {
            return cur_emitted;
        }
    }
    return cur_emitted; // exceeded recursion
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_cornell_box(Entity **elist, Entity **eworld, Camera **camera, int nx, int ny, ImageTexture** texture, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        int i = 0;
        elist[i++] = new FlipFace(new YZRect(0, 555, 0, 555, 555, new Lambertian(new ConstantTexture(Vec3(0.12, 0.45, 0.15)))));
        elist[i++] = new YZRect(0, 555, 0, 555, 0, new Lambertian(new ConstantTexture(Vec3(0.65, 0.05, 0.05))));
        elist[i++] = new XZRect(113, 443, 127, 432, 554, new DiffuseLight(new ConstantTexture(Vec3(1.0, 1.0, 1.0))));
        elist[i++] = new XZRect(0, 555, 0, 555, 0, new Lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73))));
        elist[i++] = new FlipFace(new XZRect(0, 555, 0, 555, 555, new Lambertian(new CheckerTexture(
            new ConstantTexture(Vec3(1, 1, 1)),
            new ConstantTexture(Vec3(0, 1, 0))
        ))));
        elist[i++] = new FlipFace(new XZRect(0, 555, 0, 555, 555, new Lambertian(new CheckerTexture(
            new ConstantTexture(Vec3(1, 1, 1)),
            new ConstantTexture(Vec3(0, 1, 0))
        ))));
        elist[i++] = new FlipFace(new XYRect(0, 555, 0, 555, 555, new Lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73)))));
        elist[i++] = new ConstantMedium(
            new Translate(
                new RotateY(
                    new Box(Vec3(0, 0, 0), Vec3(165, 330, 165), new Lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73)))),
                    15
                ),
                Vec3(265, 0, 295)
            ),
            0.05,
            new ConstantTexture(Vec3(0, 0, 0)),
            &local_rand_state
        );
        elist[i++] = new ConstantMedium(
            new Translate(
                new RotateY(
                    new Box(Vec3(0, 0, 0), Vec3(165, 165, 165), new Lambertian(new ConstantTexture(Vec3(0.73, 0.73, 0.73)))),
                    -18
                ),
                Vec3(130, 0, 65)
            ),
            0.01,
            new ConstantTexture(Vec3(0.8, 0.8, 0.8)),
            &local_rand_state
        );
        *eworld = new EntityList(elist, i);

        Vec3 lookfrom(278, 278, -800);
        Vec3 lookat(278, 278, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.0;
        *camera = new Camera(
            lookfrom,
            lookat,
            Vec3(0,1,0),
            40.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus,
            0.0,
            1.0
        );
    }
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int maxx, int maxy, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= maxx) || (j >= maxy)) return;
    int pixel_index = j*maxx + i;
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void texture_init(unsigned char* tex_data, int nx, int ny, ImageTexture** tex) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *tex = new ImageTexture(tex_data, nx, ny);
    }
}

__global__ void render(Vec3* fb, int max_x, int max_y, int ns, Camera **cam, Entity **world, hiprandState *randState) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = randState[pixel_index];
    Vec3 col(0,0,0);
    Vec3 background(0, 0, 0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        Ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, background, world, &local_rand_state);
    }
    randState[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

int main(int argc, char* argv[]) {
    if (argc < 5) {
        std::cerr << "Usage: " << argv[0] << " [WIDTH] [HEIGHT] [BOUNCES] [OUTPUT FILENAME]" << std::endl;
    }
    int nx = std::stoi(std::string(argv[1]));
    int ny = std::stoi(std::string(argv[2]));
    int ns = std::stoi(std::string(argv[3]));
    int tx = 16;
    int ty = 16;
    
    // Values
    int num_pixels = nx * ny;

    int tex_x, tex_y, tex_n;
    unsigned char *tex_data_host = stbi_load("assets/earthmap.jpg", &tex_x, &tex_y, &tex_n, 0);

    unsigned char *tex_data;
    checkCudaErrors(hipMallocManaged(&tex_data, tex_x * tex_y * tex_n * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(tex_data, tex_data_host, tex_x * tex_y * tex_n * sizeof(unsigned char), hipMemcpyHostToDevice));

    ImageTexture **texture;
    checkCudaErrors(hipMalloc((void **)&texture, sizeof(ImageTexture*)));
    texture_init<<<1, 1>>>(tex_data, tex_x, tex_y, texture);

    // Allocating CUDA memory
    Vec3* image;
    checkCudaErrors(hipMallocManaged((void**)&image, nx * ny * sizeof(Vec3)));

    // Allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

    // Allocate 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Building the world
    Entity **elist;
    int num_entity = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void **)&elist, num_entity * sizeof(Entity*)));
    Entity **eworld;
    checkCudaErrors(hipMalloc((void **)&eworld, sizeof(Entity*)));
    Camera **camera;
    checkCudaErrors(hipMalloc((void **)&camera, sizeof(Camera*)));
    create_cornell_box<<<1, 1>>>(elist, eworld, camera, nx, ny, texture, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(image, nx, ny,  ns, camera, eworld, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    uint8_t* imageHost = new uint8_t[nx * ny * 3 * sizeof(uint8_t)];
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * nx + i;
            imageHost[(ny - j - 1) * nx * 3 + i * 3] = 255.99 * image[pixel_index].r();
            imageHost[(ny - j - 1) * nx * 3 + i * 3 + 1] = 255.99 * image[pixel_index].g();
            imageHost[(ny - j - 1) * nx * 3 + i * 3 + 2] = 255.99 * image[pixel_index].b();
        }
    }
    stbi_write_png(argv[4], nx, ny, 3, imageHost, nx * 3);

    // Clean up
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(eworld));
    checkCudaErrors(hipFree(elist));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(image));
}